// Source: http://web.mit.edu/pocky/www/cudaworkshop/MonteCarlo/Pi.cu

// Written by Barry Wilkinson, UNC-Charlotte. Pi.cu  December 22, 2010.
//Derived somewhat from code developed by Patrick Rogers, UNC-C
//
//How to run?
//===========
//
//Single precision :
//
//nvcc -O3 pi-curand.cu ; ./a.out <thread_num>
//
//Double precision
//
//nvcc -O3 -D DP pi-curand.cu ; ./a.out <thread_num>

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <pthread.h>

#define TRIALS_PER_THREAD 4096
#define BLOCKS 256
#define THREADS 256


//Help code for switching between Single Precision and Double Precision
#ifdef DP
	typedef double Real;
	#define PI  3.14159265358979323846  // known value of pi
#else
	typedef float Real;
	#define PI 3.1415926535  // known value of pi
#endif


/**
A random number generator. 
Guidance from from http://stackoverflow.com/a/3067387/1281089
**/
Real randNumGen(){

   int random_value = rand(); //Generate a random number   
   Real unit_random = random_value / (Real) RAND_MAX; //make it between 0 and 1 
   return unit_random;
}

//struct for parameter passing between pthread calls
struct pthread_arg_struct {
    int tid;
    int total_threads;
    double total_tasks;
};


/**
The task allocated to a thread
**/
void *doCalcs(void *arguments)
{
	struct pthread_arg_struct *args = (struct pthread_arg_struct *)arguments;

	double total_threads = args -> total_threads;
	
	double total_tasks=args -> total_tasks; //total number of tasks
   int tid = args -> tid;       //obtain the value of thread id
   // printf("tid %d %lf\n", tid, total_tasks);

   //using malloc for the return variable in order make
   //sure that it is not destroyed once the thread call is finished
   double *in_count = (double *)malloc(sizeof(double));
   *in_count=0;
   
   //get the total number of iterations for a thread
   double tot_iterations= total_tasks/total_threads;
   // printf("%lf\n", tot_iterations);
   
   long counter=0;
   
   //calculation
   for(counter=0;counter<tot_iterations;counter++){
      Real x = randNumGen();
      Real y = randNumGen();
      
      Real result = sqrt((x*x) + (y*y));
      
      if(result<1){
         *in_count+=1;         //check if the generated value is inside a unit circle
      }
      
   }
   
   //get the remaining iterations calculated by thread 0
   if(tid==0){
      // int remainder = total_tasks%total_threads;
      double remainder = fmod((double)total_tasks,total_threads);
      // printf("%lf remainder\n", remainder );
      
      for(counter=0;counter<remainder;counter++){
      Real x = rand_r((unsigned int*) &tid) / (Real) RAND_MAX;
      Real y = rand_r((unsigned int*) &tid) / (Real) RAND_MAX;
      
      Real result = sqrt((x*x) + (y*y));
      
      if(result<1){
         *in_count+=1;         //check if the generated value is inside a unit circle
      }
      
   }
   }


   // printf("In count from #%d : %lf\n",tid,*in_count);
   pthread_exit((void *)in_count);     //return the in count
}



__global__ void gpu_monte_carlo(Real *estimate, hiprandState *states, int trials) {
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int points_in_circle = 0;
	Real x, y;

	hiprand_init(1234, tid, 0, &states[tid]);  // 	Initialize CURAND


	for(int i = 0; i < trials; i++) {
		x = hiprand_uniform (&states[tid]);
		y = hiprand_uniform (&states[tid]);
		points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
	}
	estimate[tid] = 4.0f * points_in_circle / (Real) trials; // return estimate of pi
}

Real host_monte_carlo(long trials) {
	Real x, y;
	long points_in_circle;
	for(long i = 0; i < trials; i++) {
		x = rand() / (Real) RAND_MAX;
		y = rand() / (Real) RAND_MAX;
		points_in_circle += (x*x + y*y <= 1.0f);
	}
  // printf("Serial- points_in_circle : %ld\n", points_in_circle);
  // printf("Serial- trials: %ld\n",trials );
	return 4.0f * points_in_circle / trials;
}

int main (int argc, char *argv[]) {
	clock_t start, stop;

	//get the total number of pthreads
	int total_threads=atoi(argv[1]);

	pthread_t threads[total_threads];
   	int rc;
   	long t;
   	void *status;
   	double tot_in=0;
   	long total_tasks=pow(2,28);

   	int trials_per_thread= total_tasks/(BLOCKS*THREADS);

	Real host[BLOCKS * THREADS];
	Real *dev;
	hiprandState *devStates;

	printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n", trials_per_thread,
BLOCKS, THREADS);

	start = clock();

	hipMalloc((void **) &dev, BLOCKS * THREADS * sizeof(Real)); // allocate device mem. for counts
	
	hipMalloc( (void **)&devStates, THREADS * BLOCKS * sizeof(hiprandState) );

	gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev, devStates,trials_per_thread);

	hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(Real), hipMemcpyDeviceToHost); // return results 

	Real pi_gpu;
	for(int i = 0; i < BLOCKS * THREADS; i++) {
		pi_gpu += host[i];
	}

	pi_gpu /= (BLOCKS * THREADS);

	stop = clock();

	// #ifdef DP
	// 	printf("GPU pi calculated in %20.18f s.\n", (stop-start)/(Real)CLOCKS_PER_SEC);

	// #else
		printf("GPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);

	// #endif
	

	// PThreads
	start = clock();
	for(t=0;t<total_threads;t++){
		struct pthread_arg_struct* args=(struct pthread_arg_struct*)malloc(sizeof *args);
		args->total_threads=total_threads;
		args->tid=t;
		// args->total_tasks=BLOCKS*THREADS*TRIALS_PER_THREAD;
    args->total_tasks=total_tasks;
     	rc = pthread_create(&threads[t], NULL, doCalcs, (void *)args);
     	if (rc){
       		printf("ERROR; return code from pthread_create() is %d\n", rc);
       		exit(-1);
       	}
    }

  	//join the threads
   	for(t=0;t<total_threads;t++){
           
      pthread_join(threads[t], &status);
	    tot_in+=*(double*)status;            //keep track of the total in count 
      // printf("tot_in IN LOOP: %lf\n", tot_in);  
     }
   // printf("tot_in : %lf\n", tot_in);
   // printf("total_tasks : %ld\n", total_tasks);  

   Real pthread_pi=4*(tot_in/total_tasks);
   stop = clock();
 //   #ifdef DP
	// 	printf("PThreads pi calculated in %20.18f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);

	// #else
		printf("PThreads pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);

	// #endif
   //End of PThreads 
	

	start = clock();
	Real pi_cpu = host_monte_carlo(total_tasks);
	stop = clock();

	// #ifdef DP
	// 	printf("CPU pi calculated in %20.18f s.\n", (stop-start)/(Real)CLOCKS_PER_SEC);

	// #else
		printf("CPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC);

	// #endif

	
	#ifdef DP
		printf("CUDA estimate of PI = %20.18f [error of %20.18f]\n", pi_gpu, pi_gpu - PI);
		printf("CPU estimate of PI = %20.18f [error of %20.18f]\n", pi_cpu, pi_cpu - PI);
		printf("PThread estimate of PI = %20.18f [error of %20.18f]\n",pthread_pi,pthread_pi - PI);

	#else
		printf("CUDA estimate of PI = %f [error of %f]\n", pi_gpu, pi_gpu - PI);
		printf("CPU estimate of PI = %f [error of %f]\n", pi_cpu, pi_cpu - PI);
		printf("PThread estimate of PI = %f [error of %f]\n",pthread_pi,pthread_pi - PI);

	#endif
	/* Last thing that main() should do */
   // pthread_exit(NULL);

   return 0;
}
